#include "srrf_cuda.h"
#include "srrf_kernel.h"
hipEvent_t start_total, stop_total;
float milliseconds = 0;
int width, height, n, border, nRingCoordinates;
double magnification, spatialRadius;
int magnification_int, widthM, heightM, outputPixNum, borderM, widthMBorderless, heightMBorderless; 
double * pixels_dev, * GxArray, * GyArray, * radArray, * SrrfArray_dev;
double * xRingCoordinates0, * yRingCoordinates0, * cubic_x, * cubic_x2, * cubic_y;

void srrf_setup(    
    const int width_in,
    const int height_in,
    const int n_in, 
    const int border_in, 
    const int nRingCoordinates_in,
    const double magnification_in,
    const double spatialRadius_in
){
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventRecord(start_total);
    width = width_in;
    height=height_in;
    n = n_in;
    border = border_in;
    nRingCoordinates = nRingCoordinates_in;
    magnification = magnification_in;
    spatialRadius = spatialRadius_in;
    magnification_int = static_cast<int>(magnification);
    widthM = width * magnification_int;
    heightM = height * magnification_int;
    outputPixNum = widthM*heightM;
    borderM = border * magnification_int;
    widthMBorderless = widthM - borderM * 2;
    heightMBorderless = heightM - borderM * 2;
    hipMalloc((void**)&pixels_dev, sizeof(double)*n*width*height);
    hipMalloc((void**)&GxArray, sizeof(double)*n*width*height);
    hipMalloc((void**)&GyArray, sizeof(double)*n*width*height);
    hipMalloc((void**)&radArray, sizeof(double)*n*widthMBorderless*heightMBorderless);
    hipMalloc((void**)&SrrfArray_dev, sizeof(double)*widthM*heightM);
    hipMalloc((void**)&xRingCoordinates0, sizeof(double)*nRingCoordinates);
    hipMalloc((void**)&yRingCoordinates0, sizeof(double)*nRingCoordinates);
    hipMalloc((void**)&cubic_x, sizeof(double)*magnification_int*nRingCoordinates*4);
    hipMalloc((void**)&cubic_x2, sizeof(double)*magnification_int*4);
    hipMalloc((void**)&cubic_y, sizeof(double)*magnification_int*nRingCoordinates*4);
    const dim3 nthrds(magnification_int, 4, 4);
    const dim3 nblcks(1, (int)nRingCoordinates/4, 1); 
    init_RingCoordinates_cubic<<<nblcks,nthrds>>>(xRingCoordinates0,yRingCoordinates0,cubic_x,cubic_x2,cubic_y,spatialRadius,magnification,borderM,nRingCoordinates);
}

void srrf_preprocess(
    const double * pixels,
    double * outputPic,    
    const int blockx,
    const int blocky,
    const int gridx,
    const int gridy,
    const int gridz
){
    hipMemcpy(pixels_dev, pixels, sizeof(double)*n*width*height, hipMemcpyHostToDevice);   
    const dim3 nthrds(blockx, blocky, 1);
    const dim3 nblcks(gridx, gridy, gridz); 
    calculateSRRF<<<nblcks, nthrds>>>(SrrfArray_dev,radArray,pixels_dev,cubic_x2,cubic_y,width,height,n,border,magnification);
    hipMemcpy(outputPic, SrrfArray_dev, sizeof(double)*widthM*heightM, hipMemcpyDeviceToHost);   

}

void srrf_end(double* time){
    hipFree(pixels_dev);
    hipFree(GxArray);
    hipFree(GyArray);
    hipFree(radArray);
    hipFree(SrrfArray_dev);
    milliseconds = 0;
    hipEventSynchronize(stop_total);
    hipEventElapsedTime(&milliseconds, start_total, stop_total);
    //std::cout << "total time : " << milliseconds << std::endl;
    *time = (double) milliseconds;
}
